#include "hip/hip_runtime.h"
#include <clocker.h>
#include <iostream>

BENCHINIT();

__global__ void add(int *i1, int *i2, int *o) { // o[] = i1[] + i2[]
    BENCHBEGIN();

    int tid = threadIdx.x;
    o[tid] = i1[tid] + i2[tid];

    o[tid] *= 2;

    o[tid] -= i1[tid];
    o[tid] -= i2[tid];

    BENCHEND();
}

int main() {
    const int size = 512;
    int a[size], c[size];
    int *d_i1, *d_i2, *d_o;

    hipMalloc(&d_i1, sizeof(int) * size);
    hipMalloc(&d_i2, sizeof(int) * size);
    hipMalloc(&d_o, sizeof(int) * size);

    for (int i = 0; i < size; i++) {
      a[i] = i + 1;
      c[i] = 0;
    }

    hipMemcpy(d_i1, a, sizeof(int) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_i2, a, sizeof(int) * size, hipMemcpyHostToDevice);

    add<<<1, size>>>(d_i1, d_i2, d_o);

    hipMemcpy(c, d_o, sizeof(int) * size, hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i++) {
      if (c[i] != (2 * (i + 1))) {
        std::cout << "Failed Verification for::" << i << " - " << c[i] << " - "
                  << (2 * (i + 1)) << " - " << a[i] << std::endl;
      }
    }

    hipFree(d_i1);
    hipFree(d_i2);
    hipFree(d_o);

    PRINTDATA();
}
